#include "hip/hip_runtime.h"
#include "../Inputs/hip/hip_runtime.h"

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fclangir \
// RUN:            -x cuda -emit-cir -target-sdk-version=12.3 \
// RUN:            %s -o %t.cir
// RUN: FileCheck --check-prefix=CIR-HOST --input-file=%t.cir %s

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fclangir \
// RUN:            -fcuda-is-device -emit-cir -target-sdk-version=12.3 \
// RUN:            %s -o %t.cir
// RUN: FileCheck --check-prefix=CIR-DEVICE --input-file=%t.cir %s

// Attribute for global_fn
// CIR-HOST: [[Kernel:#[a-zA-Z_0-9]+]] = {{.*}}#cir.cu.kernel_name<_Z9global_fni>{{.*}}

__host__ void host_fn(int *a, int *b, int *c) {}
// CIR-HOST: cir.func @_Z7host_fnPiS_S_
// CIR-DEVICE-NOT: cir.func @_Z7host_fnPiS_S_

__device__ void device_fn(int* a, double b, float c) {}
// CIR-HOST-NOT: cir.func @_Z9device_fnPidf
// CIR-DEVICE: cir.func @_Z9device_fnPidf

__global__ void global_fn(int a) {}
// CIR-DEVICE: @_Z9global_fni({{.*}} cc(ptx_kernel)
// LLVM-DEVICE: define dso_local ptx_kernel void @_Z9global_fni

// Check for device stub emission.

// CIR-HOST: @_Z24__device_stub__global_fni{{.*}}extra([[Kernel]])
// CIR-HOST: %[[#CIRKernelArgs:]] = cir.alloca {{.*}}"kernel_args"
// CIR-HOST: %[[#Decayed:]] = cir.cast(array_to_ptrdecay, %[[#CIRKernelArgs]]
// CIR-HOST: cir.call @__cudaPopCallConfiguration
// CIR-HOST: cir.get_global @_Z24__device_stub__global_fni
// CIR-HOST: cir.call @hipLaunchKernel

// LLVM-HOST: void @_Z24__device_stub__global_fni
// LLVM-HOST: %[[#KernelArgs:]] = alloca [1 x ptr], i64 1, align 16
// LLVM-HOST: %[[#GEP1:]] = getelementptr ptr, ptr %[[#KernelArgs]], i32 0
// LLVM-HOST: %[[#GEP2:]] = getelementptr ptr, ptr %[[#GEP1]], i64 0
// LLVM-HOST: call i32 @__cudaPopCallConfiguration
// LLVM-HOST: call i32 @hipLaunchKernel(ptr @_Z24__device_stub__global_fni

int main() {
  global_fn<<<1, 1>>>(1);
}
// CIR-DEVICE-NOT: cir.func @main()

// CIR-HOST: cir.func @main()
// CIR-HOST: cir.call @_ZN4dim3C1Ejjj
// CIR-HOST: cir.call @_ZN4dim3C1Ejjj
// CIR-HOST: [[Push:%[0-9]+]] = cir.call @__cudaPushCallConfiguration
// CIR-HOST: [[ConfigOK:%[0-9]+]] = cir.cast(int_to_bool, [[Push]]
// CIR-HOST: cir.if [[ConfigOK]] {
// CIR-HOST: } else {
// CIR-HOST:   [[Arg:%[0-9]+]] = cir.const #cir.int<1>
// CIR-HOST:   cir.call @_Z24__device_stub__global_fni([[Arg]])
// CIR-HOST: }

// LLVM-HOST: define dso_local i32 @main
// LLVM-HOST: alloca %struct.dim3
// LLVM-HOST: alloca %struct.dim3
// LLVM-HOST: call void @_ZN4dim3C1Ejjj
// LLVM-HOST: call void @_ZN4dim3C1Ejjj
// LLVM-HOST: [[LLVMConfigOK:%[0-9]+]] = call i32 @__cudaPushCallConfiguration
// LLVM-HOST: br [[LLVMConfigOK]], label %[[#Good:]], label [[#Bad:]]
// LLVM-HOST: [[#Good]]:
// LLVM-HOST:   br label [[#End:]]
// LLVM-HOST: [[#Bad]]:
// LLVM-HOST:   call void @_Z24__device_stub__global_fni
// LLVM-HOST:   br label [[#End]]
