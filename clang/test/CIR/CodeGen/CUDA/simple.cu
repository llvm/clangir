#include "hip/hip_runtime.h"
#include "../Inputs/hip/hip_runtime.h"

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fclangir \
// RUN:            -x cuda -emit-cir %s -o %t.cir
// RUN: FileCheck --check-prefix=CIR-HOST --input-file=%t.cir %s

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fclangir \
// RUN:            -fcuda-is-device -emit-cir %s -o %t.cir
// RUN: FileCheck --check-prefix=CIR-DEVICE --input-file=%t.cir %s

// Attribute for global_fn
// CIR-HOST: [[Kernel:#[a-zA-Z_0-9]+]] = {{.*}}#cir.cuda_kernel_name<_Z9global_fnv>{{.*}}

__host__ void host_fn(int *a, int *b, int *c) {}
// CIR-HOST: cir.func @_Z7host_fnPiS_S_
// CIR-DEVICE-NOT: cir.func @_Z7host_fnPiS_S_

__device__ void device_fn(int* a, double b, float c) {}
// CIR-HOST-NOT: cir.func @_Z9device_fnPidf
// CIR-DEVICE: cir.func @_Z9device_fnPidf

#ifdef __CUDA_ARCH__
__global__ void global_fn() {}
#else
__global__ void global_fn();
#endif
// CIR-HOST: @_Z24__device_stub__global_fnv(){{.*}}extra([[Kernel]])
// CIR-DEVICE: @_Z9global_fnv

// Make sure `global_fn` indeed gets emitted
__host__ void x() { auto v = global_fn; }
