
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang_cc1 -fclangir -std=c++11 -fcuda-is-device -triple nvptx64-nvidia-cuda -emit-llvm -o - %s | FileCheck --check-prefix=DEVICE-LLVM %s
// RUN: %clang_cc1 -fclangir -std=c++11 -fcuda-is-device -triple nvptx64-nvidia-cuda -emit-cir -o - %s | FileCheck --check-prefix=DEVICE-CIR %s
// RUN: echo "GPU binary would be here" > %t

struct textureReference {
  int desc;
};

enum ReadMode {
  ElementType = 0,
  NormalizedFloat = 1
};

template <typename T, int dim = 1, enum ReadMode mode = ElementType>
struct __attribute__((device_builtin_texture_type)) texture : public textureReference {
};

texture<float, 2, NormalizedFloat> tex;

// DEVICE-LLVM: @tex = addrspace(1) externally_initialized global i64 undef, align 4
// DEVICE-CIR: cir.global external addrspace(offload_global) @tex = #cir.undef : !s64i {alignment = 4 : i64, cu.externally_initialized = #cir.cu.externally_initialized}
