#include "hip/hip_runtime.h"
#include "../Inputs/hip/hip_runtime.h"

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fclangir \
// RUN:            -fcuda-is-device -emit-cir -target-sdk-version=12.3 \
// RUN:            %s -o %t.cir
// RUN: FileCheck --check-prefix=CIR --input-file=%t.cir %s

__global__ void fn() {
  int i = 0;
  __shared__ int j;
  j = i;
}

// CIR: cir.global "private" internal dsolocal addrspace(offload_local) @_ZZ2fnvE1j : !s32i
// CIR: cir.func @_Z2fnv
// CIR: [[Local:%[0-9]+]] = cir.alloca !s32i, !cir.ptr<!s32i>, ["i", init]
// CIR: [[Shared:%[0-9]+]] = cir.get_global @_ZZ2fnvE1j : !cir.ptr<!s32i, addrspace(offload_local)>
// CIR: [[Tmp:%[0-9]+]] = cir.load [[Local]] : !cir.ptr<!s32i>, !s32i
// CIR: cir.store [[Tmp]], [[Shared]] : !s32i, !cir.ptr<!s32i, addrspace(offload_local)>
