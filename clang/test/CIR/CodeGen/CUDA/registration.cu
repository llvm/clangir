#include "../Inputs/hip/hip_runtime.h"

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fclangir \
// RUN:            -x cuda -emit-cir -target-sdk-version=12.3 \
// RUN:            -fcuda-include-gpubinary fatbin.o\
// RUN:            %s -o %t.cir
// RUN: FileCheck --check-prefix=CIR-HOST --input-file=%t.cir %s

// CIR-HOST: module @"{{.*}}" attributes{{.*}}cir.cu.binary_handle = #cir.cu.binary_handle<fatbin.o>{{.*}}
