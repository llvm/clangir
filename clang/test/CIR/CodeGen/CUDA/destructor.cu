#include "hip/hip_runtime.h"
#include "../Inputs/hip/hip_runtime.h"

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fclangir \
// RUN:            -x cuda -emit-cir -target-sdk-version=12.3 \
// RUN:            %s -o %t.cir
// RUN: FileCheck --check-prefix=CIR-HOST --input-file=%t.cir %s

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fclangir \
// RUN:            -fcuda-is-device -emit-cir -target-sdk-version=12.3 \
// RUN:            %s -o %t.cir
// RUN: FileCheck --check-prefix=CIR-DEVICE --input-file=%t.cir %s

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fclangir \
// RUN:            -fcuda-is-device -emit-llvm -target-sdk-version=12.3 \
// RUN:            %s -o %t.ll
// RUN: FileCheck --check-prefix=LLVM-DEVICE --input-file=%t.ll %s

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fclangir \
// RUN:            -x cuda -emit-llvm -target-sdk-version=12.3 \
// RUN:            %s -o %t.ll
// RUN: FileCheck --check-prefix=LLVM-HOST --input-file=%t.ll %s

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda \
// RUN:            -fcuda-is-device -emit-llvm -target-sdk-version=12.3 \
// RUN:            %s -o %t.ll
// RUN: FileCheck --check-prefix=OGCG-DEVICE --input-file=%t.ll %s

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu \
// RUN:            -x cuda -emit-llvm -target-sdk-version=12.3 \
// RUN:            %s -o %t.ll
// RUN: FileCheck --check-prefix=OGCG-HOST --input-file=%t.ll %s

// Make sure we do emit device-side kernel even if it's only referenced
// by the destructor of a variable not present on device.
template<typename T> __global__ void f(T) {}
template<typename T> struct A {
  ~A() { f<<<1, 1>>>(T()); }
};

// CIR-HOST: module
// CIR-DEVICE: module
// CIR-DEVICE: cir.func dso_local @_Z1fIiEvT_
// LLVM-DEVICE: define dso_local ptx_kernel void @_Z1fIiEvT_
// OGCG-DEVICE: define ptx_kernel void @_Z1fIiEvT_

// CIR-HOST: cir.func {{.*}} @_ZN1AIiED2Ev{{.*}} {
// CIR-HOST:   cir.call @__cudaPushCallConfiguration
// CIR-HOST:   cir.call @_Z16__device_stub__fIiEvT_
// CIR-HOST: }

// LLVM-HOST: define linkonce_odr void @_ZN1AIiED2Ev
// LLVM-HOST: call i32 @__cudaPushCallConfiguration(
// LLVM-HOST: call void @_Z16__device_stub__fIiEvT_

// OGCG-HOST: define linkonce_odr void @_ZN1AIiED2Ev
// OGCG-HOST: call i32 @__cudaPushCallConfiguration(
// OGCG-HOST: call void @_Z16__device_stub__fIiEvT_



A<int> a;
