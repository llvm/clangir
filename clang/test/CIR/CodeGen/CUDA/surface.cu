
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang_cc1 -fclangir -std=c++11 -fcuda-is-device -triple nvptx64-nvidia-cuda -emit-llvm -o - %s | FileCheck --check-prefix=DEVICE-LLVM %s
// RUN: %clang_cc1 -fclangir -std=c++11 -fcuda-is-device -triple nvptx64-nvidia-cuda -emit-cir -o - %s | FileCheck --check-prefix=DEVICE-CIR %s
// RUN: echo "GPU binary would be here" > %t
// RUN: %clang_cc1 -fclangir -std=c++11 -triple x86_64-unknown-linux-gnu -target-sdk-version=8.0 -fcuda-include-gpubinary %t -emit-llvm -o - %s | FileCheck --check-prefix=HOST %s

struct surfaceReference {
  int desc;
};

template <typename T, int dim = 1>
struct __attribute__((device_builtin_surface_type)) surface : public surfaceReference {
};

// Partial specialization over `void`.
template<int dim>
struct __attribute__((device_builtin_surface_type)) surface<void, dim> : public surfaceReference {
};

surface<void, 2> surf;

// DEVICE-LLVM: @surf = addrspace(1) externally_initialized global i64 undef, align 4
// DEVICE-CIR: cir.global external addrspace(offload_global) @surf = #cir.undef : !s64i {alignment = 4 : i64, cu.externally_initialized = #cir.cu.externally_initialized}
// HOST: @surf = global %"struct.surface<void, 2>" zeroinitializer, align 4